#include "hip/hip_runtime.h"


#include "timing.c"
#include "gpuMerge.h"

#define UPB 100
#define LEVELS 5
#define MAXDRET 102400
#define THRESH 2

float * fillArray(int n, int upbound)
{
   int i;

   float *ret = (float *)malloc(sizeof(float) * n );

   /* Intializes random number generator */
   //seeds the random number generator used by the function rand.
    srand(1);//time(NULL));

   /* generate n random numbers from 0 to unbound - 1 */
   for( i = 0 ; i < n ; i++ ) {
      ret[i] = rand() % upbound * 1.0f;
   }

   return ret;
}

void printArray(float *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%5.0f ", arr[i]);

   printf("\n");
}

long cpuReduce(float *h_in, int n)
{
    long total =0;

	int i;
    for(i = 0; i < n; i ++)
        total += h_in[i];

	//printf("Total found by  cpu == %ld\n", total);

    return (float)total;
}





void usage()
{
   printf("Usage: ./progName blockWidth numElementsInput p \n");
}


int gpuMain(int argc, const char **argv)
{
   // create a large workload so we can easily measure the
   // performance difference on CPU and GPU

   // to run this program: ./a.out blockWidth numElements p
   int shouldPrint = 0;
   if(argc < 3 || argc > 4) {
      usage();
      return 1;
   } else  if(argc == 3){
         shouldPrint = 0;
   } else if(argv[3][0]=='p'){
         shouldPrint=1;
   } else {
         usage();
         return 1;
   }

   //
   int tile_width = atoi(argv[1]);

   if ( ! tile_width )
   {
       printf("Wrong argument passed in for blockWidth!\n");
       exit(-1);
   }
   int n = atoi(argv[2]); //size of 1D input array

   if ( ! n )
   {
       printf("Wrong argument passed in for size of input array!\n");
       exit(-1);
   }

   // set up host memory
   float *h_in, *h_out, *d_in, *d_out;

   //int sizeDout[LEVELS]; //we can have at most 5 levels of kernel launch

   h_out = (float *)malloc(MAXDRET * sizeof(float));

   memset(h_out, 0, MAXDRET * sizeof(float));

   //generate input data from random generator
   h_in = fillArray(n, UPB);

   if( ! h_in || ! h_out )
   {
       printf("Error in host memory allocation!\n");
       exit(-1);
   }

   int num_block = ceil(n / (float)tile_width);
   dim3 block(tile_width, 1, 1);
   dim3 grid(num_block, 1, 1);

   // allocate storage for the device
   hipMalloc((void**)&d_in, sizeof(float) * n);
   hipMalloc((void**)&d_out, sizeof(float) * MAXDRET);
   hipMemset(d_out, 0, sizeof(float) * MAXDRET);

   // copy input to the device
   hipMemcpy(d_in, h_in, sizeof(float) * n, hipMemcpyHostToDevice);

   // time the kernel launches using CUDA events
   hipEvent_t launch_begin, launch_end;
   hipEventCreate(&launch_begin);
   hipEventCreate(&launch_end);

   printf("The input array is:\n");
   //print out original array
   if(shouldPrint)
       printArray(h_in, n);


   //----------------------time many kernel launches and take the average time--------------------
   const size_t num_launches = 10;
   float average_simple_time = 0;
   int num_in = n, num_out = ceil((float)n / tile_width);
   int launch = 1;

   printf("Timing simple GPU implementation… \n");
   for(int i = 0; i < num_launches; ++i)
   {
       // record a CUDA event immediately before and after the kernel launch
       hipEventRecord(launch_begin,0);
       //gpuSummationReduce<<<grid, block, tile_width * sizeof(float)>>>(d_in, d_out, n);
       //hipMemcpy(h_out, d_out, sizeof(float) * num_block, hipMemcpyDeviceToHost);
       while( 1 )
       {
           if(launch % 2 == 1) // odd launch
               gpuSummationReduce<<<grid, block, tile_width * sizeof(float)>>>(d_in, d_out, num_in);
           else
               gpuSummationReduce<<< grid, block, tile_width * sizeof(float) >>>(d_out, d_in, num_in);

           hipDeviceSynchronize();

           // if the number of local max returned by kernel is greater than the threshold,
           // we do reduction on GPU for these returned local maxes for another pass,
           // until, num_out < threshold
           if(num_out >= THRESH)
           {
               num_in = num_out;
               num_out = ceil((float)num_out / tile_width);
               grid.x = num_out; //change the grid dimension in x direction
               //hipMemset(d_in, 0, n * sizeof(float));//reset d_in, used for output of next iteration
           }
           else
           {
               //copy the ouput of last lauch back to host,
               if(launch % 2 == 1)
                  hipMemcpy(h_out, d_out, sizeof(float) * num_out, hipMemcpyDeviceToHost);
               else
                  hipMemcpy(h_out, d_in, sizeof(float) * num_out, hipMemcpyDeviceToHost);

               break;
           }
           launch ++;
       }//end of while

       hipEventRecord(launch_end,0);
       hipEventSynchronize(launch_end);

       // measure the time spent in the kernel
       float time = 0;
       hipEventElapsedTime(&time, launch_begin, launch_end);

       average_simple_time += time;
  }
  average_simple_time /= num_launches;
  printf(" done! GPU time cost in second: %f\n", average_simple_time / 1000);

  printf("The output array from device is:\n");
  //if(shouldPrint)
      printArray(h_out, num_out);


  //------------------------ now time the sequential code on CPU------------------------------

  // time many multiplication calls and take the average time
  float average_cpu_time = 0;
  clock_t now, then;
  int num_cpu_test = 3;
  long cpuTotal = 0;

  printf("Timing CPU implementation…\n");

  for(int i = 0; i < num_cpu_test; ++i)
  {
    // timing on CPU
    then = clock();
    cpuTotal = cpuReduce(h_in, n);
    now = clock();

    // measure the time spent on CPU
    float time = 0;
    time = timeCost(then, now);

    average_cpu_time += time;
  }
  average_cpu_time /= num_cpu_test;
  printf(" done. CPU time cost in second: %f\n", average_cpu_time);

  //if (shouldPrint)
      printf("CPU finding total is %ld\n", cpuTotal);

  //--------------------------------clean up-----------------------------------------------------
  hipEventDestroy(launch_begin);
  hipEventDestroy(launch_end);

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);

  free(h_in);
  free(h_out);

  return 0;
}

