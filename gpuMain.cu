#include "hip/hip_runtime.h"


#include "timing.c"
#include "gpuMerge.h"

#define UPB 100
#define LEVELS 5
#define MAXDRET 102400
#define THRESH 2

int * fillArray(int n, int upbound)
{
   int i;

   int *ret = (int *)malloc(sizeof(int) * n );

   /* Intializes random number generator */
   //seeds the random number generator used by the function rand.
    srand(1);//time(NULL));

   /* generate n random numbers from 0 to unbound - 1 */
   for( i = 0 ; i < n ; i++ ) {
      ret[i] = i;//rand() % upbound * 1.0f;
   }

   return ret;
}

void printArray(int *arr, int n){

   int i;

   for(i = 0; i < n; i ++)
      printf("%d ", arr[i]);

   printf("\n");
}

int cpuReduce(int *h_in, int n)
{
    int total = 0;

	int i;
    for(i = 0; i < n; i ++)
        total += h_in[i];

	//printf("Total found by  cpu == %d\n", total);

    return total;
}

void usage()
{
   printf("Usage: ./progName blockWidth numElementsInput p \n");
}


int gpuMain(int argc, const char **argv)
{
   // create a large workload so we can easily measure the
   // performance difference on CPU and GPU

   // to run this program: ./a.out blockWidth numElements p
   int shouldPrint = 0;
   if(argc < 3 || argc > 4) {
      usage();
      return 1;
   } else  if(argc == 3){
         shouldPrint = 0;
   } else if(argv[3][0]=='p'){
         shouldPrint=1;
   } else {
         usage();
         return 1;
   }

   //
   float tile_width = atoi(argv[1]);

   if ( ! tile_width )
   {
       printf("Wrong argument passed in for blockWidth!\n");
       exit(-1);
   }
   int n = atoi(argv[2]); //size of 1D input array

   if ( ! n )
   {
       printf("Wrong argument passed in for size of input array!\n");
       exit(-1);
   }

   // set up host memory
   int *h_in, *h_out, *d_in, *d_out;

   //int sizeDout[LEVELS]; //we can have at most 5 levels of kernel launch

   h_out = (int *)malloc(MAXDRET * sizeof(int));

   memset(h_out, 0, MAXDRET * sizeof(int));

   //generate input data from random generator
   h_in = fillArray(n, UPB);

   if( ! h_in || ! h_out )
   {
       printf("Error in host memory allocation!\n");
       exit(-1);
   }

   float num_block = ceil(n / (float)tile_width);
   dim3 block(tile_width, 1, 1);
   dim3 grid(num_block, 1, 1);

   // allocate storage for the device
   hipMalloc((void**)&d_in, sizeof(int) * n);
   hipMalloc((void**)&d_out, sizeof(int) * MAXDRET);
   hipMemset(d_out, 0, sizeof(int) * MAXDRET);

   // copy input to the device
   hipMemcpy(d_in, h_in, sizeof(int) * n, hipMemcpyHostToDevice);

   // time the kernel launches using CUDA events
   hipEvent_t launch_begin, launch_end;
   hipEventCreate(&launch_begin);
   hipEventCreate(&launch_end);

   printf("The input array is:\n");
   //print out original array
   if(shouldPrint)
       printArray(h_in, n);


   //----------------------time many kernel launches and take the average time--------------------
   
   float average_simple_time = 0;
   int num_in = n, num_out = ceil((float)n / tile_width);
   int launch = 1;

   printf("Timing simple GPU implementation… \n");
   
       // record a CUDA event immediately before and after the kernel launch
       hipEventRecord(launch_begin,0);
    



       while( 1 )
       {
           if(launch % 2 == 1) // odd launch
               gpuSummationReduce<<<grid, block, tile_width * sizeof(float)>>>(d_in, d_out, num_in);
           else
               gpuSummationReduce<<< grid, block, tile_width * sizeof(float) >>>(d_out, d_in, num_in);

           hipDeviceSynchronize();

           // if the number of local max returned by kernel is greater than the threshold,
           // we do reduction on GPU for these returned local maxes for another pass,
           // until, num_out < threshold
           if(num_out >= THRESH)
           {
               num_in = num_out;
               num_out = ceil((float)num_out / tile_width);
               grid.x = num_out; //change the grid dimension in x direction
               //hipMemset(d_in, 0, n * sizeof(int));//reset d_in, used for output of next iteration
           }
           else
           {
               //copy the ouput of last lauch back to host,
               if(launch % 2 == 1)
                  hipMemcpy(h_out, d_out, sizeof(int) * num_out, hipMemcpyDeviceToHost);
               else
                  hipMemcpy(h_out, d_in, sizeof(int) * num_out, hipMemcpyDeviceToHost);

               break;
           }
           launch ++;
       }//end of while





       hipEventRecord(launch_end,0);
       hipEventSynchronize(launch_end);

       // measure the time spent in the kernel
       float time = 0;
       hipEventElapsedTime(&time, launch_begin, launch_end);

       average_simple_time += time;
 
 
  printf(" done! GPU time cost in second: %f\n", average_simple_time / 1000);
  printf(" done! GPU time cost in second: %f\n", time / 1000);

  printf("The output array from device is:\n");
  //if(shouldPrint)
      printArray(h_out, num_out);


  //------------------------ now time the sequential code on CPU------------------------------

  // time many multiplication calls and take the average time
  float average_cpu_time = 0;
  clock_t now, then;

  long cpuTotal = 0;

  printf("Timing CPU implementation…\n");

 
    // timing on CPU
    then = clock();
    cpuTotal = cpuReduce(h_in, n);
    now = clock();

    // measure the time spent on CPU
   time = 0;
    time = timeCost(then, now);

    average_cpu_time += time;
 
  //average_cpu_time /= num_cpu_test;
  printf(" done. CPU time cost in second: %f\n", average_cpu_time);
  printf(" done. CPU time cost in second: %f\n", time);

  //if (shouldPrint)
      printf("CPU finding total is %ld\n", cpuTotal);

  //--------------------------------clean up-----------------------------------------------------
  hipEventDestroy(launch_begin);
  hipEventDestroy(launch_end);

  // deallocate device memory
  hipFree(d_in);
  hipFree(d_out);

  free(h_in);
  free(h_out);

  return 0;
}

